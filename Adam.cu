#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <math.h>
#include <string>
#include <eigen3/Eigen/Dense>
#include "preprocessing.h"
#include "optimizers.h"

std::vector<float> Adam(float alpha, float beta1, float beta2, Eigen::MatrixXf Train, Eigen::MatrixXf labels)
{
    std::vector<float> params {0, 0};
    int n_param = 2;
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(1,1);
    float tolerance = 0.05;
    Eigen::MatrixXf m(n_param, 1);
    Eigen::MatrixXf v(n_param, 1);
    for(int row = 0; row != m.rows(); row++)
    {
        for(int col = 0; col != m.cols(); col++)
        {
            m(row, col) = 0;
        }
    }
    for(int row = 0; row != v.rows(); row++)
    {
        for(int col = 0; col != v.cols(); col++)
        {
            v(row, col) = 0;
        }
    }
    Eigen::MatrixXf m_hat = m;
    Eigen::MatrixXf v_hat = v;
    float l = distribution(generator);
    float sigma = distribution(generator);
    int maximum_iterations = 1000;
    int iteration = 0;
    float epsilon = pow(10, -8);
    Eigen::MatrixXf gradient = GetGradient(Train, labels, sigma, l);
    Eigen::MatrixXf gradient_2(2,1);
    while(!(std::abs(gradient(0,0)) <= tolerance && std::abs(gradient(1,0)) <= tolerance) && !(iteration >= maximum_iterations))
    {
        iteration += 1;
        std::cout << "iteration number " + std::to_string(iteration) << std::endl;
        gradient = GetGradient(Train, labels, sigma, l);
        for(int row = 0; row != gradient_2.rows(); row++)
        {
            for(int col = 0; col != gradient_2.cols(); col++)
            {
                gradient_2(row, col) = gradient(row, col)*gradient(row, col);
            }
        }
        std::cout << "dsigma " + std::to_string(gradient(0,0))<< std::endl;
        std::cout << "dl " + std::to_string(gradient(1,0))  << std::endl;
        m = beta1 * m + (1 - beta1) * gradient;
        v = beta2 * v + (1 - beta2) * gradient_2;
        m_hat = m/(1 - pow(beta1, iteration));
        v_hat = v/(1 - pow(beta2, iteration));
        sigma = sigma  - alpha * m_hat(0,0) / (sqrt(v_hat(0,0) + epsilon));
        l = l  - alpha * m_hat(1,0) / (sqrt(v_hat(1,0) + epsilon));

    }
    params[0] = sigma;
    params[1] = l;
    
    return params;
    
}
