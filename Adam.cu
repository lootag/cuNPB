#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <math.h>
#include <eigen3/Eigen/Dense>
#include "preprocessing.h"
#include "optimizers.h"

std::vector<float> Adam(float alpha, float beta1, float beta2, Eigen::MatrixXf Train, Eigen::MatrixXf labels)
{
    std::vector<float> params {0, 0};
    int n_param = 2;
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(1,1);
    float tolerance = 0.0001;
    Eigen::MatrixXf m(n_param, 1);
    Eigen::MatrixXf v(n_param, 1);
    for(int row = 0; row != m.rows(); row++)
    {
        for(int col = 0; col != m.cols(); col++)
        {
            m(row, col) = 0;
        }
    }
    for(int row = 0; row != v.rows(); row++)
    {
        for(int col = 0; col != v.cols(); col++)
        {
            v(row, col) = 0;
        }
    }
    Eigen::MatrixXf m_hat = m;
    Eigen::MatrixXf v_hat = v;
    float l = distribution(generator);
    float sigma = distribution(generator);
    int maximum_iterations = 1000;
    int iteration = 0;
    float epsilon = pow(10, -8);
    Eigen::MatrixXf gradient = GetGradient(Train, labels, sigma, l);
    Eigen::MatrixXf gradient_2(2,1);
    bool converged = gradient[0] == tolerance && gradient[1] == tolerance;
    bool out_of_bounds = iteration > maximum_iterations;
    while(!converged && !out_of_bounds)
    {
        iteration += 1;
        gradient = GetGradient(Train, labels, sigma, l);
        for(int row = 0; row != gradient_2.rows(); row++)
        {
            for(int col = 0; col != gradient_2.cols(); col++)
            {
                gradient_2(row, col) = gradient(row, col)*gradient(row, col);
            }
        }
        std::cout << "dsigma" << std::endl;
        std::cout <<  gradient(0,0) << std::endl;
        std::cout << "dl" << std::endl;
        std::cout << gradient(1,0) << std::endl;
        m = beta1 * m + (1 - beta1) * gradient;
        v = beta2 * v + (1 - beta2) * gradient_2;
        m_hat = m/(1 - pow(beta1, iteration));
        v_hat = v/(1 - pow(beta2, iteration));
        sigma = sigma  - alpha * m_hat(0,0) / (sqrt(v_hat(0,0) + epsilon));
        l = l  - alpha * m_hat(1,0) / (sqrt(v_hat(1,0) + epsilon));

    }
    params[0] = sigma;
    params[1] = l;
    
    return params;
    
}
