#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <math.h>
#include <string>
#include <eigen3/Eigen/Dense>
#include "preprocessing.h"
#include "optimizers.h"

std::vector<float> Adam(float alpha, float beta1, float beta2, Eigen::MatrixXf Train, Eigen::MatrixXf labels, float tolerance, int maximum_iterations)
{
    std::vector<float> params {0, 0};
    int n_param = 2;
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(1,1);
    Eigen::MatrixXf m(n_param, 1);
    Eigen::MatrixXf v(n_param, 1);
    for(int row = 0; row != m.rows(); row++)
    {
        for(int col = 0; col != m.cols(); col++)
        {
            m(row, col) = 0;
        }
    }
    for(int row = 0; row != v.rows(); row++)
    {
        for(int col = 0; col != v.cols(); col++)
        {
            v(row, col) = 0;
        }
    }
    Eigen::MatrixXf m_hat = m;
    Eigen::MatrixXf v_hat = v;
    float l = distribution(generator);
    float sigma = distribution(generator);
    int iteration = 0;
    float epsilon = pow(10, -8);
    Eigen::MatrixXf gradient = GetGradient(Train, labels, sigma, l);
    Eigen::MatrixXf gradient_2(2,1);
    float percentage_change = 1000;
    while((!(percentage_change <= tolerance) && !(iteration >= maximum_iterations)) || iteration <= 1000)
    {
        iteration += 1;
        std::cout << "iteration number " + std::to_string(iteration) << std::endl;
        gradient = GetGradient(Train, labels, sigma, l);
        for(int row = 0; row != gradient_2.rows(); row++)
        {
            for(int col = 0; col != gradient_2.cols(); col++)
            {
                gradient_2(row, col) = gradient(row, col)*gradient(row, col);
            }
        }
        m = beta1 * m + (1 - beta1) * gradient;
        v = beta2 * v + (1 - beta2) * gradient_2;
        m_hat = m/(1 - pow(beta1, iteration));
        v_hat = v/(1 - pow(beta2, iteration));
        float sum_this = l + sigma;
        sigma = sigma  - alpha * m_hat(0,0) / (sqrt(v_hat(0,0) + epsilon));
        l = l  - alpha * m_hat(1,0) / (sqrt(v_hat(1,0) + epsilon));
        float sum_next = l + sigma;
        percentage_change = std::abs(sum_next - sum_this)/sum_this * 100;
        std::cout << "Variation of " + std::to_string(percentage_change) + " %" << std::endl; 

    }
    params[0] = sigma;
    params[1] = l;
    
    return params;
    
}
