#include "hip/hip_runtime.h"
#include <iostream>
#include <eigen3/Eigen/Dense>
#include <math.h>
#include "model.h"
#include "preprocessing.h"
#include "enumerators.h"

void model::PredictVariance()
{
    Eigen::MatrixXf X_Train = model::get_X_Train();
    Eigen::MatrixXf Y_Train = model::get_Y_Train();
    Eigen::MatrixXf X_Test = model::get_Y_Test();
    float sigma = model::get_sigma();
    float l = model::get_l();
    
    kernel_type type_standard = standard;
    Eigen::MatrixXf K_Train_Train = GetKernel(X_Train, X_Train, sigma, l, type_standard);
    Eigen::MatrixXf K_Test_Test = GetKernel(X_Test, X_Test, sigma, l, type_standard);
    Eigen::MatrixXf K_Train_Test = GetKernel(X_Train, X_Test, sigma, l, type_standard);
    Eigen::MatrixXf K_Test_Train = GetKernel(X_Test, X_Train, sigma, l, type_standard);
    Eigen::MatrixXf I;
    I.setIdentity(K_Train_Train.rows(), K_Train_Train.cols());
    
    Eigen::MatrixXf MiddleTerm = K_Train_Train + (pow(sigma, 2) * I);
    Eigen::MatrixXf MiddleTerm_inv = MiddleTerm.inverse();
    Eigen::MatrixXf RH = Multiply(K_Test_Train, MiddleTerm_inv.transpose());
    RH = Multiply(RH, K_Train_Test);
    Eigen::MatrixXf Variance = K_Test_Test - RH + (pow(sigma, 2) * I);
    model::set_Sigma_2(Variance);
}