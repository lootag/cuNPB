#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <math.h>
#include <eigen3/Eigen/Dense>

using std::vector;

__global__ void getKernel(const float *a, const float *b, float *c, int cols) 
{
  // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
    if(row < cols && col < cols)
    {
        c[col + row * cols] = 0;
        for (int k = 0; k < cols; k++) 
        {
            c[col + row * cols] += (a[k + row * cols] - b[col + k * cols]) * (a[k + row * cols] - b[col + k * cols]);
        }
    }
  
  
}



int main() {
  // Matrix size of 4 x 3;
  int rows = 4;
  int cols = 3;
  Eigen::MatrixXf A(rows,cols);
  Eigen::MatrixXf C(rows,rows);
  int tmp = 1;
  for(int row = 0; row != A.rows(); row++)
  {
      for(int col = 0; col != A.cols(); col++)
      {
          A(row, col) = tmp;
          tmp += 1;
      }
  }

  

  // Size (in bytes) of matrix
  size_t bytes_a = rows * cols * sizeof(float);
  size_t bytes_b = cols * rows * sizeof(float);
  size_t bytes_c = rows * rows * sizeof(float);

  // Host vectors
  vector<float> h_a(rows * cols);
  vector<float> h_b(cols * rows);
  vector<float> h_c(rows * rows);

  for(int row = 0; row != A.rows(); row++)
  {
      for(int col = 0; col != A.cols(); col++)
      {
          h_a[col + row*cols] = A(row,col);
      }
  }

  for(int row = 0; row != A.transpose().rows(); row++)
  {
      for(int col = 0; col != A.transpose().cols(); col++)
      {
          h_b[col + row*A.transpose().cols()] = A.transpose()(row,col);
      }
  }

 


  


  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes_b, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = 1;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  getKernel<<<blocks, threads>>>(d_a, d_b, d_c, rows);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  for(int row = 0; row != C.rows(); row++)
  {
      for(int col = 0; col != C.cols(); col++)
      {
          C(row,col) = h_c[col + row*C.cols()];
      }
  }

  for(int row = 0; row != C.rows(); row++)
  {
      for(int col = 0; col != C.cols(); col++)
      {
          std::cout << C(row,col) << std::endl;
      }
  }


  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}