#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <math.h>
#include "preprocessing.h"
#include <eigen3/Eigen/Dense>

using std::vector;

__global__ void getKernel(const float *a, const float *b, float *c, int rows, int cols, int rowsB) 
{
  // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
    if(row < rows && col < rowsB)
    {
        c[col + row * rowsB] = 0;
        for (int k = 0; k < cols; k++) 
        {
            c[col + row * rowsB] += (a[k + row * cols] - b[col + k * rowsB]) * (a[k + row * cols] - b[col + k * rowsB]);
        }
    }
  
  
}



Eigen::MatrixXf GetKernel(Eigen::MatrixXf A, Eigen::MatrixXf B) 
{
  // Matrix size of 4 x 3;
  int rows = A.rows();
  int cols = A.cols();
  int rowsB = B.rows();
  Eigen::MatrixXf C(rows,rowsB);
  

  

  // Size (in bytes) of matrix
  size_t bytes_a = rows * cols * sizeof(float);
  size_t bytes_b = cols * rowsB * sizeof(float);
  size_t bytes_c = rows * rowsB * sizeof(float);

  // Host vectors
  vector<float> h_a(rows * cols);
  vector<float> h_b(cols * rowsB);
  vector<float> h_c(rows * rowsB);

  for(int row = 0; row != A.rows(); row++)
  {
      for(int col = 0; col != A.cols(); col++)
      {
          h_a[col + row*cols] = A(row,col);
      }
  }

  for(int row = 0; row != B.transpose().rows(); row++)
  {
      for(int col = 0; col != B.transpose().cols(); col++)
      {
          h_b[col + row * B.transpose().cols()] = B.transpose()(row,col);
      }
  }

 


  


  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes_b, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  float fTHREADS = THREADS;
  float frows = rows;
  float fBLOCKS = ceil(frows/fTHREADS);
  int BLOCKS = fBLOCKS;
  

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  getKernel<<<blocks, threads>>>(d_a, d_b, d_c, rows, cols, rowsB);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  for(int row = 0; row != C.rows(); row++)
  {
      for(int col = 0; col != C.cols(); col++)
      {
          C(row,col) = h_c[col + row*C.cols()];
      }
  }




  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return C;
}