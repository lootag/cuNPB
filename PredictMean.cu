#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "model.h"
#include "preprocessing.h"
#include "enumerators.h"
#include <eigen3/Eigen/dense>

void model::PredictMean()
{
    kernel_type type_standard = standard;
    Eigen::MatrixXf X_Train = model::get_X_Train();
    Eigen::MatrixXf Y_Train = model::get_Y_Train();
    Eigen::MatrixXf X_Test = model::get_X_Test();
    float sigma = model::get_sigma();
    float l = model::get_l();
    Eigen::MatrixXf K_Test_Train = GetKernel(X_Test, X_Train, sigma, l, type_standard);
    Eigen::MatrixXf K_Train_Train = GetKernel(X_Train, X_Train, sigma, l, type_standard);
    Eigen::MatrixXf MiddleTerm = K_Train_Train + (pow(sigma, 2) * Eigen::MatrixXf::Identity(K_Train_Train.rows(), K_Train_Train.col())); 
    Eigen::MatrixXf MiddleTerm_inv = MiddleTerm.inverse();
    Eigen::MatrixXf mu = Multiply(K_Test_Train, MiddleTerm_inv.transpose());
    mu = Multiply(mu, X_Train.transpose());
    model::set_mu(mu);

}