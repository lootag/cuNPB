// This program calculates matrix multiplication (SGEMM) using cuBLAS
// By: Nick from CoffeeBeforeArch

#include <hipblas.h>
#include <hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include "./MatrixMultiply.h"



bool MatrixMultiply(float * featureM, float * featureN, float * result, 
  int count_m, int count_n, int size, int gpu_id) {
  float *dev_featureM = 0;
  float *dev_featureN = 0;
  float *dev_result = 0;
  const float alpha = 1, beta = 0;
  hipblasHandle_t handle;
  hipError_t cudaStatus;

  cudaStatus = hipSetDevice(gpu_id);
  if (cudaStatus != hipSuccess) {
      printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
      goto out;
  }
  hipblasCreate(&handle);

  cudaStatus = hipMalloc((void**)&dev_featureM, count_m * size * sizeof(float));
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMalloc failed!\n", __func__, __LINE__);
      goto out;
  }
  cudaStatus = hipMalloc((void**)&dev_featureN, count_n * size * sizeof(float));
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMalloc failed!\n", __func__, __LINE__);
      goto out;
  }
  cudaStatus = hipMalloc((void**)&dev_result, count_m * count_n * sizeof(float));
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMalloc failed!\n", __func__, __LINE__);
      goto out;
  }

  cudaStatus = hipMemcpy(dev_featureM, featureM, count_m * size * sizeof(float), 
      hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMalloc failed!\n", __func__, __LINE__);
      goto out;
  }
  cudaStatus = hipMemcpy(dev_featureN, featureN, count_n * size * sizeof(float), 
      hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMalloc failed!\n", __func__, __LINE__);
      goto out;
  }

  /*

  CUBLAS assumes that the matrix in the device is stored in column major:

  " where α and β are scalars, and A , B and C are matrices stored in column-major 
  format with dimensions op ( A ) m × k , op ( B ) k × n and C m × n , respectively. 

   Also, for matrix A


   // Multiply the arrays A and B on GPU and save the result in C (coloum-major)
    // C(m,n) = A(m,k) * B(k,n)

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);
   */

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, count_n, count_m, size, 
      &alpha, dev_featureN, size, dev_featureM, size, &beta, dev_result, count_n);
  cudaStatus = hipDeviceSynchronize();

  cudaStatus = hipMemcpy(result, dev_result, count_m * count_n  * sizeof(float), 
      hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
      printf("%s, line %d, hipMemcpy failed!\n", __func__, __LINE__);
      goto out;
  }

out:
  if(dev_featureM) hipFree(dev_featureM);
  if(dev_featureN) hipFree(dev_featureN);
  if(dev_result) hipFree(dev_result);
  hipblasDestroy(handle);
  return cudaStatus == hipSuccess;
}